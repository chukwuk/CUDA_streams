#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <cmath>
#include <string>
#include <hip/hip_runtime.h>

#include "reductionsum.h"

#define IDX2C(i,j,ld) (((i)*(ld))+(j))

#define c(x) #x
#define stringify(x) c(x)

#define t(s1,s2) s1##s2
#define tg(s1,s2) t(s1,s2)

#define tgg(s1,s2,s3) tg(tg(s1,s2),s3)
#define tggg(s1,s2,s3,s4) tg(tgg(s1,s2,s3),s4)


using namespace std;

inline
hipError_t checkCudaErrors(hipError_t result, string functioncall = "")
{
//#if defined(DEBUG) || defined(_DEBUG)
  //fprintf(stderr, "CUDA Runtime Error: %d\n", result);
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error for this function call ( %s ) : %s\n", 
            functioncall.c_str(), hipGetErrorString(result));
    assert(result == hipSuccess);
  }
//#endif
  return result;
}

int
main( int argc, char* argv[ ] )
{ 
  //srand(time(0));
//  fprintf (stderr, "Amount of data transfered to the device is %lld GB\n", bytes4euc/1000000000);
 
  long int numData = 1000000000;
  long int sumNumData =  (numData+1)/2;
  double* reduceData = new double[numData];
  double* sumData = new double[sumNumData];   

  for (int i = 0; i < numData; i++) {
      reduceData[i] = (rand() % 8)/1.0; 
  }
   
  int BLOCKSIZE = 128;
  int NUMBLOCKS = (sumNumData + BLOCKSIZE-1)/BLOCKSIZE;
    
  fprintf (stderr, "NUMBER OF BLOCKS is %d\n", NUMBLOCKS);
  
 
  // allocate memory on device

  double* reduceDataDev;
  double* sumDataDev; 

  // Create CUDA events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  

  // Record the start event
  hipEventRecord(start, 0); 
  
  hipError_t status;
  long int reduceDataSize = (sizeof(double) * numData);
  fprintf (stderr, "Amount of data transfered to the device is %ld Bytes\n", reduceDataSize);
   
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&reduceDataDev), reduceDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&reduceDataDev), reduceDataSize) ");

  long int sumDataSize = (sizeof(double) * sumNumData) ;
  fprintf (stderr, "Amount of data transfered to the device is %ld Bytes\n", sumDataSize);
   
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&sumDataDev), sumDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&sumDataDev), sumDataSize); ");  

  // copy data from host memory to the device:

  status = hipMemcpy(reduceDataDev, reduceData, reduceDataSize, hipMemcpyHostToDevice );
  // checks for cuda errors
  checkCudaErrors( status,"hipMemcpy(reduceDataDev, reduceData, reduceDataSize, hipMemcpyHostToDevice );");  
 

  // allocate number of threads in a block  
  dim3 threads(BLOCKSIZE, 1, 1 );

  // allocate number of blocks
  dim3 grid(NUMBLOCKS, 1, 1 );
  
 
  
  // call the kernel
  reductionSum<<< grid, threads >>>( reduceDataDev, sumDataDev, numData);
  
  status = hipDeviceSynchronize( );
  
   
  checkCudaErrors( status," reductionSum<<< grid, threads >>>( reduceDataDev, sumDataDev, numData); ");
 
  status = hipGetLastError(); 
  
  checkCudaErrors( status,"hipGetLastError()");  

  // copy data device memory to host:
  hipMemcpy(sumData, sumDataDev,  sumDataSize , hipMemcpyDeviceToHost);  
  // checks for cuda errors
  checkCudaErrors( status, " hipMemcpy(sumData, sumDataDev,  sumDataSize , hipMemcpyDeviceToHost);"); 
  
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // Calculate elapsed time
  float GpuTime = 0;
  hipEventElapsedTime(&GpuTime, start, stop); 
  
  printf("  GPU time: %f milliseconds\n", GpuTime);

  printf(" summation values: %f \n", reduceData[(numData-2)]);
  printf(" summation values: %f \n", reduceData[(numData-1)]);
  printf(" summation values: %f \n", sumData[(sumNumData-1)]); 
  
  double sum = 0.00;
  
  
  // Record the start event
  hipEventRecord(start, 0); 
     
  for (int i = 0; i < numData; i++) {
       sum+=reduceData[i]; 
  }
  
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // Calculate elapsed time
  hipEventElapsedTime(&GpuTime, start, stop); 
  printf("  GPU time: %f milliseconds\n", GpuTime);
  
  // free device memory 
  hipFree( sumDataDev );
  hipFree( reduceDataDev );
  delete[] sumData;
  delete[] reduceData;
  

  return 0;
};	
