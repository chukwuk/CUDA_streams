#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <cmath>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>


#include "reductionsum.h"


#ifndef NUMDATA

 #define NUMDATA 3000000000

#endif

#define IDX2C(i,j,ld) (((i)*(ld))+(j))

#define c(x) #x
#define stringify(x) c(x)

#define t(s1,s2) s1##s2
#define tg(s1,s2) t(s1,s2)

#define tgg(s1,s2,s3) tg(tg(s1,s2),s3)
#define tggg(s1,s2,s3,s4) tg(tgg(s1,s2,s3),s4)




using namespace std;


inline
hipError_t checkCudaErrors(hipError_t result, string functioncall = "")
{
//#if defined(DEBUG) || defined(_DEBUG)
  //fprintf(stderr, "CUDA Runtime Error: %d\n", result);
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error for this function call ( %s ) : %s\n", 
            functioncall.c_str(), hipGetErrorString(result));
    assert(result == hipSuccess);
  }
//#endif
  return result;
}

int
main( int argc, char* argv[ ] )
{ 
   
  unsigned long int numData;

  if (argc > 1) {
      numData = std::stoi(argv[1]);
  
  } else {

       numData = NUMDATA;
  }
  
  fprintf (stderr, "NUMBER OF DATA is %lu\n", numData);

  unsigned int nCols = 6;

  unsigned long int sumNumData =  numData/nCols;
  
  int* reduceData;
  int* sumData; 
   
  hipError_t status;
  unsigned long int reduceDataSize = (sizeof(int) * numData);
  fprintf (stderr, "Amount of reduceData data transfered to the device is %lu GB\n", reduceDataSize/1000000000);
  
  unsigned long int sumDataSize = (sizeof(int) * sumNumData) ;
  fprintf (stderr, "Amount of sumData data transfered to the device is %lu GB\n", sumDataSize/1000000000);
  
  // pinned data
  
  hipHostMalloc((void**)&reduceData, reduceDataSize);
  hipHostMalloc((void**)&sumData, sumDataSize);

  
  //memset(reduceData, 1, reduceDataSize); 
  
  for (unsigned int i = 0; i < numData; i++) {
       reduceData[i] = 1;
  } 

 
  // allocate memory on device

  int* reduceDataDev;
  int* sumDataDev; 

  // Create CUDA events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  

  
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&reduceDataDev), reduceDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&reduceDataDev), reduceDataSize) ");
 
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&sumDataDev), sumDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&sumDataDev), sumDataSize); ");  

  
  
 
  
  // Record the start event
  hipEventRecord(start, 0); 

  // copy data from host memory to the device:

  status = hipMemcpy(reduceDataDev, reduceData, reduceDataSize, hipMemcpyHostToDevice );
  // checks for cuda errors
  checkCudaErrors( status,"hipMemcpy(reduceDataDev, reduceData, reduceDataSize, hipMemcpyHostToDevice );");  
  
  int BLOCKSIZE;
  int NUMBLOCKS;
  int MINGRIDSIZE;  
  
  
  hipOccupancyMaxPotentialBlockSize( &MINGRIDSIZE, &BLOCKSIZE, 
                                      reductionSum, 0, 0); 
   
  BLOCKSIZE = 128;
  NUMBLOCKS = (sumNumData + BLOCKSIZE-1)/BLOCKSIZE;
   
  
  // allocate number of threads in a block  
  dim3 threads(BLOCKSIZE, 1, 1 );

  // allocate number of blocks
  dim3 grid(NUMBLOCKS, 1, 1 );
  
   
      

  reductionSum<<< grid, threads >>>( reduceDataDev, sumDataDev, sumNumData, nCols, 0);

  status = hipDeviceSynchronize( );
   
  checkCudaErrors( status," reductionSum<<< grid, threads >>>( reduceDataDev, sumDataDev, numData, sumNumData); ");
 
  status = hipGetLastError(); 
  
  checkCudaErrors( status,"hipGetLastError()");   


   // copy data from device memory to host 
  hipMemcpy(sumData, sumDataDev, sumDataSize, hipMemcpyDeviceToHost);  
  // checks for cuda errors
  checkCudaErrors( status, " hipMemcpy(sumData, sumDataDev,  sumDataSize , hipMemcpyDeviceToHost);"); 
  
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // Calculate elapsed time
  float GpuTime = 0;
  hipEventElapsedTime(&GpuTime, start, stop); 
  
  printf("Time for sequential transfer and execute: %f milliseconds\n", GpuTime);

  printf(" summation values: %i \n", sumData[0]); 
  

  for (int i = 0; i < sumNumData; i++) {
      if (sumData[i] - 6 != 0) {
         printf(" The value that is wrong is: %i, %i\n",i, sumData[i]);
	 break; 
      }  
  }

  hipFree( sumDataDev );
  hipFree( reduceDataDev );
   
  hipHostFree( sumData );
  hipHostFree( reduceData );

  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devId);
  printf("Device : %s\n", prop.name);
  hipSetDevice(devId); 


   
  unsigned long int nStreams = 4;
  unsigned long int streamSize = numData / nStreams;
  unsigned long int streamBytes = streamSize * sizeof(int); 
  unsigned long int streamSizeResult = sumNumData / nStreams;
  unsigned long int streamBytesResult = streamSizeResult * sizeof(int);
 // const int bytes = n * sizeof(int);
  

   // create streams
  hipStream_t stream[nStreams];
  for (int i = 0; i < nStreams; ++i) {
    hipStreamCreate(&stream[i]);
  }
   
  int* reduceStrData;
  int* sumStrData; 
  
  
  // pinned data
  
  hipHostMalloc((void**)&reduceStrData, reduceDataSize);
  hipHostMalloc((void**)&sumStrData, sumDataSize);

    
  //memset(reduceData, 1, reduceDataSize); 
  
  for (unsigned int i = 0; i < numData; i++) {
       reduceStrData[i] = 1;
  } 

    
  int* reduceStrDataDev;
  int* sumStrDataDev; 
  
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&reduceStrDataDev), reduceDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&reduceStrDataDev), reduceDataSize) ");
 
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&sumStrDataDev), sumDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&sumStrDataDev), sumDataSize); ");  



  

  // Record the start event
  hipEventRecord(start, 0); 

  
  NUMBLOCKS = (streamSizeResult + BLOCKSIZE-1)/BLOCKSIZE;
  grid.x = NUMBLOCKS;
  for (int i = 0; i < nStreams; ++i) { 
    unsigned long int offset = i * streamSize; 
    int offsetResult = i * streamSizeResult;
    hipMemcpyAsync(&reduceStrDataDev[offset], &reduceStrData[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);  
    reductionSum<<<grid, threads, 0, stream[i]>>>( reduceStrDataDev, sumStrDataDev, streamSizeResult, nCols, offsetResult);
    hipMemcpyAsync(&sumStrData[offsetResult], &sumStrDataDev[offsetResult], streamBytesResult, hipMemcpyDeviceToHost, stream[i]);
  }
  
  
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // Calculate elapsed time
  
  hipEventElapsedTime(&GpuTime, start, stop); 
  
  printf("Time for asynchronous V1 transfer and execute (ms): %f milliseconds\n", GpuTime);
  printf(" summation values: %i \n", sumStrData[0]); 
 
  for (int i = 0; i < sumNumData; i++) {
      if (sumStrData[i] - 6 != 0) {
         printf(" The value that is wrong is: %i, %i\n",i, sumStrData[i]);
	 break; 
      }  
  }
  
  
  hipFree( sumStrDataDev );
  hipFree( reduceStrDataDev );
   
  hipHostFree( sumStrData );
  hipHostFree( reduceStrData );


  // second asynchronous  transfer 
  int* reduceStrOneData;
  int* sumStrOneData; 
  
  
  // pinned data
  
  hipHostMalloc((void**)&reduceStrOneData, reduceDataSize);
  hipHostMalloc((void**)&sumStrOneData, sumDataSize);

    
  //memset(reduceData, 1, reduceDataSize); 
  
  for (unsigned int i = 0; i < numData; i++) {
       reduceStrOneData[i] = 1;
  } 

    
  int* reduceStrOneDataDev;
  int* sumStrOneDataDev; 
  
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&reduceStrOneDataDev), reduceDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&reduceStrDataDev), reduceDataSize) ");
 
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&sumStrOneDataDev), sumDataSize);
  // checks for cuda errors  
  checkCudaErrors( status, " hipMalloc( (void **)(&sumStrDataDev), sumDataSize); ");  
 

  // Record the start event
  hipEventRecord(start, 0); 

    
  for (int i = 0; i < nStreams; ++i) { 
    unsigned long int offset = i * streamSize;
    hipMemcpyAsync(&reduceStrOneDataDev[offset], &reduceStrOneData[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);  
  }
  

   
  for (int i = 0; i < nStreams; ++i) { 
    int offsetResult = i * streamSizeResult;
    reductionSum<<<grid, threads, 0, stream[i]>>>( reduceStrOneDataDev, sumStrOneDataDev, streamSizeResult, nCols, offsetResult);
  }
  

  
  for (int i = 0; i < nStreams; ++i) { 
    int offsetResult = i * streamSizeResult;
    hipMemcpyAsync(&sumStrOneData[offsetResult], &sumStrOneDataDev[offsetResult], streamBytesResult, hipMemcpyDeviceToHost, stream[i]);
  }
  
    
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // Calculate elapsed time
  
  hipEventElapsedTime(&GpuTime, start, stop); 
  
  printf("Time for asynchronous V2 transfer and execute (ms): %f milliseconds\n", GpuTime);
  printf(" summation values: %i \n", sumStrOneData[0]); 
 
  for (int i = 0; i < sumNumData; i++) {
      if (sumStrOneData[i] - 6 != 0) {
         printf(" The value that is wrong is: %i, %i\n",i, sumStrOneData[i]);
	 break; 
      }  
  }
  
  
  hipFree( sumStrOneDataDev );
  hipFree( reduceStrOneDataDev );
   
  hipHostFree( sumStrOneData );
  hipHostFree( reduceStrOneData );



  
  /*
  double sum = 0.00;
   
  // Record the start event
  hipEventRecord(start, 0); 
     
  for (int i = 0; i < numData; i++) {
       sum+=1.0; 
  }


  //printf("  CPU summation: %f \n", sum);
  
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // Calculate elapsed time
  hipEventElapsedTime(&GpuTime, start, stop); 

  printf("  CPU time: %f milliseconds\n", GpuTime);
  
    
  // free device memory 
  hipFree( sumDataDev );
  hipFree( reduceDataDev );
  hipHostFree(sumData);
  hipHostFree(reduceData);
  */
  return 0;
};	
